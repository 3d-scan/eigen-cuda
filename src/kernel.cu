
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include <kernel.hpp>
#include <Eigen/Core>

// CUDA Version
namespace Kernel
{
    __global__ void cu_dot(Eigen::Vector3d *a, Eigen::Vector3d *b, double *out, size_t N)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if(idx < N)
        {
            int i = 3 * idx;
            out[i]   += a[i].dot(b[i]);
        }
        return;
    }

    // TODO: memcpy and malloc
    double dot(std::vector<Eigen::Vector3d> v1, std::vector<Eigen::Vector3d> v2)
    {
        double* ret = new double[v1.size()];
        int n = v1.size();

        cu_dot<<<(n+1023)/1024, 1024>>>(v1.data(), v2.data(), ret, n);
        
        return ret[0];
    }
}

#endif