
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

///// Eigen Workaround Stuff
// See http://eigen.tuxfamily.org/dox/TopicCUDA.html
// and http://eigen.tuxfamily.org/index.php?title=3.3#Experimental_CUDA_support
#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
// workaround issue between gcc >= 4.7 and cuda 5.5
#if (defined __GNUC__) && (__GNUC__>4 || __GNUC_MINOR__>=7)
  #undef _GLIBCXX_ATOMIC_BUILTINS
  #undef _GLIBCXX_USE_INT128
#endif
/////////////////////////////

#include <kernel.hpp>
#include <Eigen/Core>

#include <iostream>
#include <stdio.h>


static void HandleError( cudaError_t err, const char *file, int line ) {
	// CUDA error handeling from the "CUDA by example" book
	if (err != cudaSuccess) {
		printf( "%s in %s at line %d\n", cudaGetErrorString( err ),
				file, line );
		exit( EXIT_FAILURE );
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

// CUDA Version
namespace Kernel
{
    // TODO: Get this working
    __global__ void cu_dot(Eigen::Vector3d *v1, Eigen::Vector3d *v2, double *out, size_t N)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if(idx < N)
        {
            out[idx] = v1[idx].dot(v2[idx]);
        }
        return;
        // int idx = blockIdx.x * blockDim.x + threadIdx.x;
        // if(idx < N)
        // {
        //     out[idx] = 33.0;
        // }
    }

    // The wrapper for the calling of the actual kernel
    double dot(std::vector<Eigen::Vector3d> v1, std::vector<Eigen::Vector3d> v2)
    {        
        int n = v1.size();
        double *ret = new double[n];

        // Allocate device arrays
        Eigen::Vector3d *dev_v1, *dev_v2;
        HANDLE_ERROR(cudaMalloc((void **)&dev_v1, sizeof(Eigen::Vector3d)*n));
        HANDLE_ERROR(cudaMalloc((void **)&dev_v2, sizeof(Eigen::Vector3d)*n));
        double* dev_ret;
        HANDLE_ERROR(cudaMalloc((void **)&dev_ret, sizeof(double)*n));

        // Copy to device
        HANDLE_ERROR(cudaMemcpy(dev_v1, v1.data(), sizeof(Eigen::Vector3d)*n, cudaMemcpyHostToDevice));
        HANDLE_ERROR(cudaMemcpy(dev_v2, v2.data(), sizeof(Eigen::Vector3d)*n, cudaMemcpyHostToDevice));

        // Dot product
        cu_dot<<<(n+1023)/1024, 1024>>>(v1.data(), v2.data(), dev_ret, n);
        
        // Copy to host
        HANDLE_ERROR(cudaMemcpy(ret, dev_ret, sizeof(double)*n, cudaMemcpyDeviceToHost));

        // Return
        // std::cerr << "ret =";
        // for (int i=0; i<n; ++i)
        // {
        //     std::cerr << "  " << ret[i];
        // }
        // std::cerr << std::endl;
        return ret[0];
    }
}

#endif